#include "hip/hip_runtime.h"
#pragma once



#include "common/hip/hip_vector_types.h"
#include "common/hip/hip_runtime_api.h"

#include <assert.h>

//#define USE_SMEM
#define BLOCKSIZE (32 * 32)

float4* pCudaPos0 = NULL; float4* pCudaPos1 = NULL;
float4* pCudaPosOld0 = NULL; float4* pCudaPosOld1 = NULL;

float4* pPosIn, * pPosOut;
float4* pPosOldIn, * pPosOldOut;

int iteration = 0;

__device__ int2 NextNeigh(int n)
{
	if (n == 0)
	{
		return make_int2(-1, -1);
	}
	if (n == 1)
	{
		return make_int2(0, -1);
	}
	if (n == 2)
	{
		return make_int2(1, -1);
	}
	if (n == 3)
	{
		return make_int2(1, 0);
	}
	if (n == 4)
	{
		return make_int2(1, 1);
	}
	if (n == 5)
	{
		return make_int2(0, 1);
	}
	if (n == 6)
	{
		return make_int2(-1, 1);
	}
	if (n == 7)
	{
		return make_int2(-1, 0);
	}

	if (n == 8)
	{
		return make_int2(-2, -2);
	}
	if (n == 9)
	{
		return make_int2(2, -2);
	}
	if (n == 10)
	{
		return make_int2(2, 2);
	}
	if (n == 11)
	{
		return make_int2(-2, 2);
	}

	return make_int2(0, 0);
}

__global__ void verlet_cuda(float4* pos_vbo, float4* nor_vbo, float4* g_pos_in, float4* g_pos_old_in, float4* g_pos_out, float4* g_pos_old_out,
	int side, float stiffness, float damp, float inverse_mass, int coll_primitives)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = index % side;
	int iy = index / side;

	//g_pos[index] = make_float4(threadIdx.x, blockIdx.x, blockDim.x, index);
	//return;

#ifdef USE_SMEM
	__shared__ float4 smem_pos[BLOCKSIZE];
	__shared__ float4 smem_pos_old[BLOCKSIZE];
	__shared__ float4 smem_vel[BLOCKSIZE];

	int ix_smem = threadIdx.x % side;
	int iy_smem = threadIdx.x / side;

	smem_pos[threadIdx.x] = g_pos_in[index];
	smem_pos[threadIdx.x] = g_pos_in[index];
	smem_pos_old[threadIdx.x] = g_pos_old_out[index];

	for (int k = 0; k < 12; k++)
	{
		int2 coord = NextNeigh(k);
		int j = coord.x;
		int i = coord.y;

		if (((iy_smem + i) < 0) || ((iy_smem + i) > (side - 1)))
			continue;

		if (((ix_smem + j) < 0) || ((ix_smem + j) > (side - 1)))
			continue;

		int index_neigh_smem = (iy_smem + i) * side + ix_smem + j;
		int index_neigh = (iy + i) * side + ix + j;

		smem_pos[index_neigh_smem] = g_pos_in[index_neigh];
		smem_pos_old[index_neigh_smem] = g_pos_old_out[index_neigh];
		smem_vel[index_neigh_smem] = g_pos_in[index_neigh];
	}

	__syncthreads();

	volatile float4 posData = smem_pos[threadIdx.x];    // ensure coalesced read
	volatile float4 posOldData = smem_pos_old[threadIdx.x];
	volatile float4 velData = smem_vel[threadIdx.x];
#else
	volatile float4 posData = g_pos_in[index];    // ensure coalesced read
	volatile float4 posOldData = g_pos_old_in[index];
#endif

	float3 pos = make_float3(posData.x, posData.y, posData.z);
	float3 pos_old = make_float3(posOldData.x, posOldData.y, posOldData.z);
	float3 vel = (pos - pos_old) / 0.01;
	float3 gravity = make_float3(0.0, -9.81, 0.0);

	// used for computation of the normal
	float3 normal = make_float3(0, 0, 0);
	float3 last_diff = make_float3(0, 0, 0);
	int iters = 0;

	float3 force = make_float3(0.0, 0.0, 0.0);
	float inv_mass = inverse_mass;
	if (index <= (side - 1.0))
		inv_mass = 0.0f;

	float step = 1.0 / (side - 1.0);

	for (int k = 0; k < 12; k++)
	{
		int2 coord = NextNeigh(k);
		int j = coord.x;
		int i = coord.y;

#ifdef USE_SMEM
		if (((iy_smem + i) < 0) || ((iy_smem + i) > (side - 1)))
			continue;

		if (((ix_smem + j) < 0) || ((ix_smem + j) > (side - 1)))
			continue;

		int index_neigh_smem = (iy_smem + i) * side + ix_smem + j;

		volatile float4 pos_neighData = smem_pos[index_neigh_smem];
#else
		if (((iy + i) < 0) || ((iy + i) > (side - 1)))
			continue;

		if (((ix + j) < 0) || ((ix + j) > (side - 1)))
			continue;

		int index_neigh = (iy + i) * side + ix + j;

		volatile float4 pos_neighData = g_pos_in[index_neigh];
#endif
		float3 pos_neigh = make_float3(pos_neighData.x, pos_neighData.y, pos_neighData.z);

		float3 diff = pos_neigh - pos;

		float3 curr_diff = diff;	// curr diff is the normalized direction of the spring
		curr_diff = normalize(curr_diff);

		if ((iters > 0) && (k < 8))
		{
			float angle = dot(curr_diff, last_diff);
			if (angle > 0.0)
			{
				normal += cross(last_diff, curr_diff);
			}
		}
		last_diff = curr_diff;

		float2 fcoord = make_float2(coord) * step;
		float rest_length = length(fcoord);

		force += (curr_diff * (length(diff) - rest_length)) * stiffness - vel * damp * 0.5;
		if (k < 8)
		{
			iters += 1;
		}
	}

	normal = normalize(normal / -(iters - 1.0));

	float3 acc = make_float3(0, 0, 0);
	acc = (force + gravity) * inv_mass;

	// verlet
	float3 tmp = pos;
	pos = pos * 2 - pos_old + acc * 0.01 * 0.01;
	pos_old = tmp;

	// collision with a sphere
	if (coll_primitives & 1)
	{
		float3 center = make_float3(0.5, -0.5, 0.25);
		float radius = 0.3;

		if (length(pos - center) < radius)
		{
			// collision
			float3 coll_dir = normalize(pos - center);
			pos = center + coll_dir * radius;
		}
	}

	// collision with a plane
	if (coll_primitives & 2)
	{
		if (pos.y < -0.6)
		{
			pos.y = -0.6;
			pos_old += (pos - pos_old) * 0.03;
		}
	}

	__syncthreads();

	pos_vbo[index] = make_float4(pos.x, pos.y, pos.z, posData.w);
	nor_vbo[index] = make_float4(normal.x, normal.y, normal.z, 0.0);

	g_pos_out[index] = make_float4(pos.x, pos.y, pos.z, posData.w);
	g_pos_old_out[index] = make_float4(pos_old.x, pos_old.y, pos_old.z, posOldData.w);
}

// size is the total number of float (= number of particles * 4)
void InitCuda(const int size)
{
	const unsigned int num_threads = size / 4;
	const unsigned int mem_size = sizeof(float4) * num_threads;

	// allocate device memory for float4 version
	checkCudaErrors(hipMalloc((void**)&pCudaPos0, mem_size));	// positions
	checkCudaErrors(hipMalloc((void**)&pCudaPos1, mem_size));	// positions
	checkCudaErrors(hipMalloc((void**)&pCudaPosOld0, mem_size));	// old positions
	checkCudaErrors(hipMalloc((void**)&pCudaPosOld1, mem_size));	// old positions

	iteration = 0;
}

//Round a / b to nearest higher integer value
int iDivUp(int a, int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void ComputeGridSize(uint n, uint blockSize, uint& numBlocks, uint& numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = iDivUp(n, numThreads);
}

void UploadCuda(float* positions, float* positions_old, const int size)
{
	assert(pCudaPos0 != NULL);
	assert(pCudaPosOld0 != NULL);

	const unsigned int num_threads = size / 4;
	//cutilCondition(0 == (num_threads % 4));
	const unsigned int mem_size = sizeof(float4) * num_threads;

	// copy host memory to device
	// NOTE: it is not necessary to copy in each iteration, just swap the buffers -> huge save of computation time

	if ((iteration % 2) == 0)
	{
		pPosIn = pCudaPos0;
		pPosOut = pCudaPos1;
		pPosOldIn = pCudaPosOld0;
		pPosOldOut = pCudaPosOld1;
	}
	else
	{
		pPosIn = pCudaPos1;
		pPosOut = pCudaPos0;
		pPosOldIn = pCudaPosOld1;
		pPosOldOut = pCudaPosOld0;
	}

	if (iteration == 0)
	{
		hipMemcpy(pPosIn, positions, mem_size, hipMemcpyHostToDevice);
		hipMemcpy(pPosOldIn, positions_old, mem_size, hipMemcpyHostToDevice);
	}

	iteration++;
}

void VerletCuda(float4* pos_vbo, float4* nor_vbo, float* positions, float* positions_old, const int size, const int& side, const float& stiffness,
	const float& damp, const float& inverse_mass, const int& coll_primitives)
{
	// setup execution parameters 
	uint numThreads, numBlocks;
	uint numParticles = size / 4;

	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	// execute the kernel
	//	printf("numParticles: %d,   numThreads: %d   numBlocks: %d\n", numParticles, numThreads, numBlocks);
	verlet_cuda << < numBlocks, numThreads >> > (pos_vbo, nor_vbo, pPosIn, pPosOldIn, pPosOut, pPosOldOut, side, stiffness, damp, inverse_mass, coll_primitives);

	// stop the CPU until the kernel has been executed
	hipDeviceSynchronize();
}

void ResetCuda()
{
	// cleanup memory
	if (pCudaPos0 != NULL)
	{
		hipFree(pCudaPos0);
		hipFree(pCudaPos1);
		pCudaPos0 = NULL;
		pCudaPos1 = NULL;
	}

	if (pCudaPosOld0 != NULL)
	{
		hipFree(pCudaPosOld0);
		hipFree(pCudaPosOld1);
		pCudaPosOld0 = NULL;
		pCudaPosOld1 = NULL;
	}
}
